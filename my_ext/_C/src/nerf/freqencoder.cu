#include "hip/hip_runtime.h"
#include "util.cuh"

inline constexpr __device__ float PI() { return 3.141592653589793f; }

// inputs: [B, D]
// outputs: [B, C], C = D + D * deg * 2
__global__ void kernel_freq(
    const float* __restrict__ inputs, uint32_t B, uint32_t D, uint32_t deg, uint32_t C, float* outputs) {
  // parallel on per-element
  const uint32_t t = threadIdx.x + blockIdx.x * blockDim.x;
  if (t >= B * C) return;

  // get index
  const uint32_t b = t / C;
  const uint32_t c = t - b * C;  // t % C;

  // locate
  inputs += b * D;
  outputs += t;

  // write self
  if (c < D) {
    outputs[0] = inputs[c];
    // write freq
  } else {
    const uint32_t col      = c / D - 1;
    const uint32_t d        = c % D;
    const uint32_t freq     = col / 2;
    const float phase_shift = (col % 2) * (PI() / 2);
    outputs[0]              = __sinf(scalbnf(inputs[d], freq) + phase_shift);
  }
}

// grad: [B, C], C = D + D * deg * 2
// outputs: [B, C]
// grad_inputs: [B, D]
__global__ void kernel_freq_backward(const float* __restrict__ grad, const float* __restrict__ outputs, uint32_t B,
    uint32_t D, uint32_t deg, uint32_t C, float* grad_inputs) {
  // parallel on per-element
  const uint32_t t = threadIdx.x + blockIdx.x * blockDim.x;
  if (t >= B * D) return;

  const uint32_t b = t / D;
  const uint32_t d = t - b * D;  // t % D;

  // locate
  grad += b * C;
  outputs += b * C;
  grad_inputs += t;

  // register
  float result = grad[d];
  grad += D;
  outputs += D;

  for (uint32_t f = 0; f < deg; f++) {
    result += scalbnf(1.0f, f) * (grad[d] * outputs[D + d] - grad[D + d] * outputs[d]);
    grad += 2 * D;
    outputs += 2 * D;
  }

  // write
  grad_inputs[0] = result;
}

void freq_encode_forward(
    at::Tensor inputs, const uint32_t B, const uint32_t D, const uint32_t deg, const uint32_t C, at::Tensor outputs) {
  CHECK_CUDA(inputs);
  CHECK_CUDA(outputs);

  CHECK_CONTIGUOUS(inputs);
  CHECK_CONTIGUOUS(outputs);

  CHECK_IS_FLOATING(inputs);
  CHECK_IS_FLOATING(outputs);

  static constexpr uint32_t N_THREADS = 128;

  kernel_freq KERNEL_ARG(div_round_up(B * C, N_THREADS), N_THREADS)(
      inputs.data_ptr<float>(), B, D, deg, C, outputs.data_ptr<float>());
}

void freq_encode_backward(at::Tensor grad, at::Tensor outputs, const uint32_t B, const uint32_t D, const uint32_t deg,
    const uint32_t C, at::Tensor grad_inputs) {
  CHECK_CUDA(grad);
  CHECK_CUDA(outputs);
  CHECK_CUDA(grad_inputs);

  CHECK_CONTIGUOUS(grad);
  CHECK_CONTIGUOUS(outputs);
  CHECK_CONTIGUOUS(grad_inputs);

  CHECK_IS_FLOATING(grad);
  CHECK_IS_FLOATING(outputs);
  CHECK_IS_FLOATING(grad_inputs);

  static constexpr uint32_t N_THREADS = 128;

  kernel_freq_backward KERNEL_ARG(div_round_up(B * D, N_THREADS), N_THREADS)(
      grad.data_ptr<float>(), outputs.data_ptr<float>(), B, D, deg, C, grad_inputs.data_ptr<float>());
}

REGIST_PYTORCH_EXTENSION(nerf_freq_encode, {
  m.def("freq_encode_forward", &freq_encode_forward, "freq encode forward (CUDA)");
  m.def("freq_encode_backward", &freq_encode_backward, "freq encode backward (CUDA)");
})
